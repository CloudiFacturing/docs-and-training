// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010


#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 16;
const int blocksize = 16;

__global__
void copy(char *a, char *b)
{
        a[threadIdx.x] = b[threadIdx.x];
}

int main()
{
        char a[N] = "Hello \0\0\0\0\0\0";
        char b[N] = "World\0\0\0\0\0\0\0";

        char *ad;
        char *bd;
        const int size = N*sizeof(char);

        printf("%s\n", a);
        hipError_t err;
        err = hipMalloc( (void**)&ad, size );
	if (err != hipSuccess) {
	  printf("cudaerror: %i\n", err);
	  return EXIT_FAILURE;
	}

        hipMalloc( (void**)&bd, size );
        hipMemcpy( ad, a, size, hipMemcpyHostToDevice );

        dim3 dimBlock( blocksize, 1 );
        dim3 dimGrid( 1, 1 );
        copy<<<dimGrid, dimBlock>>>(ad, bd);
        hipMemcpy( a, ad, size, hipMemcpyDeviceToHost );
        hipFree( ad );
        hipFree( bd );

        printf("%s\n", a);
	if (strncmp(a, b, N)) {
	  printf("cuda kernel did not return expected result\n");
	  return EXIT_FAILURE;
	}
        return EXIT_SUCCESS;
}

